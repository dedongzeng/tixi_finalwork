#include <algorithm>
#include <chrono>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>


#define SIZE 1
using namespace std;

int main() {
  // freopen("swapinout.txt", "w", stdout);
  long int s[] = {
      //1,         4,          8,         16,        32,        40,
      //64,        68,         80,        128,       256,       400,
      //512,       768,        1024,      1535,      1536,      2048,
      //2264,      2348,       2600,      2888,      3020,      3164,
      //4096,      4716,       5120,      8192,      10240,     16384,
      //17920,     20000,      25600,     32768,
      37632,     42452,
      49152,     75276,      92928,     98304,     100000,    102400,
      131072,    147456,     204800,    262144,    294912,    307200,
      409860,    524288,     589824,    1000000,   1048576,   1179648,
      1228800,   1280000,    1581056,   1638400,   1638916,   2359296,
      2654208,   3538944,    4718592,   6422528,   8306688,   8388608,
      9437184,   10000000,   11075584,  11943936,  12845056,  16613376,
      16777216,  20447232,   25000000,  25690112,  35831808,  38535168,
      39321600,  49561600,   51380224,  67108864,  83886080,  121228800,
      142655492, 178438148,  186482692, 191102980, 196608000, 205520896,
      237568004, 1258291200, 3200000000};
  cout << "size,in,out" << endl;
  for (int i = 0; i < 59; i++) {
    long long size = s[i];
    void *hostArray = (void *)0;
    hipHostMalloc(&hostArray, size, hipHostMallocDefault);
    void *deviceArray = (void *)0;
    hipMalloc((void **)&deviceArray, size);

    long long a0 = std::chrono::duration_cast<std::chrono::nanoseconds>(
                       std::chrono::system_clock::now().time_since_epoch())
                       .count();
    // auto a0 = (std::chrono::system_clock::now()).time_since_epoch().count();
    hipMemcpy(deviceArray, hostArray, size, hipMemcpyHostToDevice);
    long long b0 = std::chrono::duration_cast<std::chrono::nanoseconds>(
                       std::chrono::system_clock::now().time_since_epoch())
                       .count();
    // auto b0 = (std::chrono::system_clock::now()).time_since_epoch().count();
    hipMemcpy(hostArray, deviceArray, size, hipMemcpyDeviceToHost);
    // long long c0 = std::chrono::duration_cast<std::chrono::nanoseconds>(
    //                   std::chrono::system_clock::now().time_since_epoch())
    //                   .count();
    auto c0 = (std::chrono::system_clock::now()).time_since_epoch().count();
    cout << s[i] << "," << b0 - a0 << "," << c0 - b0 << endl;
  }

  return 0;
}
